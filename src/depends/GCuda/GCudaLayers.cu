#include "hip/hip_runtime.h"
/*
  The contents of this file are dedicated by all of its authors, including

    Michael S. Gashler,
    anonymous contributors,

  to the public domain (http://creativecommons.org/publicdomain/zero/1.0/).

  Note that some moral obligations still exist in the absence of legal ones.
  For example, it would still be dishonest to deliberately misrepresent the
  origin of a work. Although we impose no legal requirements to obtain a
  license, it is beseeming for those who build on the works of others to
  give back useful improvements, or find a way to pay it forward. If
  you would like to cite us, a published paper about Waffles can be found
  at http://jmlr.org/papers/volume12/gashler11a/gashler11a.pdf. If you find
  our code to be useful, the Waffles team would love to hear how you use it.
*/

#include "GCudaLayers.h"
#include "../../GClasses/GNeuralNet.h"
#include "../../GClasses/GVec.h"

namespace GClasses {

GCudaLayer::GCudaLayer(GDomNode* pNode, GCudaEngine& engine)
: GNeuralNetLayer(), m_engine(engine)
{
	throw Ex("Sorry, GCudaLayer does not support serialization");
}

GDomNode* GCudaLayer::serialize(GDom* pDoc)
{
	throw Ex("Sorry, GNeuralNetLayerCuda does not support serialization");
	//return NULL;
}











GNeuralNetLayerCuda::GNeuralNetLayerCuda(GCudaEngine& engine, size_t inputs, size_t outputs)
: GCudaLayer(engine), m_pOutgoing(NULL)
{
	resize(inputs, outputs, NULL);
}

GNeuralNetLayerCuda::~GNeuralNetLayerCuda()
{
	delete[] m_pOutgoing;
}

void GNeuralNetLayerCuda::resize(size_t inputCount, size_t outputCount, GRand* pRand)
{
	if(inputCount == inputs() && outputCount == outputs())
		return;
	if(pRand)
		throw Ex("Sorry, GNeuralNetLayerCuda does not support preserving resizes");

	m_weights.resize(inputCount, outputCount);
	m_bias.resize(outputCount);
	m_activation.resize(outputCount);
	m_error.resize(outputCount);
	delete[] m_pOutgoing;
	m_pOutgoing = NULL;
}

// virtual
void GNeuralNetLayerCuda::resetWeights(GRand& rand)
{
	size_t inputCount = inputs();
	size_t outputCount = outputs();
	GMatrix mTmp(inputCount, outputCount);
	double mag = 1.0 / inputCount;
	for(size_t i = 0; i < inputCount; i++)
	{
		double* pW = mTmp[i];
		for(size_t j = 0; j < outputCount; j++)
			*(pW++) = rand.normal() * mag;
	}
	m_weights.upload(mTmp);
	GVec vTmp(outputCount);
	double* pB = vTmp.v;
	for(size_t i = 0; i < outputCount; i++)
		*(pB++) = rand.normal() * mag;
	m_bias.upload(vTmp.v, outputCount);
}

// virtual
void GNeuralNetLayerCuda::perturbWeights(GRand& rand, double deviation, size_t start, size_t count)
{
	// Perturb weights
	GMatrix m;
	m_weights.download(m);
	size_t n = std::min(outputs() - start, count);
	for(size_t j = 0; j < m_weights.rows(); j++)
		GVec::perturb(m[j] + start, deviation, n, rand);
	m_weights.upload(m);

	// Perturb biases
	GVec v(outputs());
	m_bias.download(v.v);
	GVec::perturb(v.v + start, deviation, n, rand);
	m_bias.upload(v.v, outputs());
}

// virtual
double* GNeuralNetLayerCuda::activation()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputs()];
	m_activation.download(m_pOutgoing);
	return m_pOutgoing;
}

// virtual
double* GNeuralNetLayerCuda::error()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputs()];
	m_error.download(m_pOutgoing);
	return m_pOutgoing;
}

// virtual
void GNeuralNetLayerCuda::copyBiasToNet()
{
	m_activation.copy(m_engine, m_bias);
	m_engine.sync();
}

// virtual
void GNeuralNetLayerCuda::feedIn(const double* pIn, size_t inputStart, size_t inputCount)
{
	m_incoming.upload(pIn, inputs());
	m_weights.feedIn(m_engine, m_incoming, m_activation, inputStart);
	m_engine.sync();
}

// virtual
void GNeuralNetLayerCuda::feedIn(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.feedIn(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), m_activation, inputStart);
		m_engine.sync();
	}
	else
		feedIn(pUpStreamLayer->activation(), inputStart, pUpStreamLayer->outputs());
}

// virtual
void GNeuralNetLayerCuda::activate()
{
	m_activation.activateTanh(m_engine);
	m_engine.sync();
}

void GNeuralNetLayerCuda::computeError(const double* pTarget)
{
	m_error.upload(pTarget, outputs());
	m_error.add(m_engine, m_activation, -1.0);
	m_engine.sync();
}

void GNeuralNetLayerCuda::deactivateError()
{
	m_error.deactivateTanh(m_engine, m_activation);
	m_engine.sync();
}

void GNeuralNetLayerCuda::backPropError(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.backPropError(m_engine, m_error, ((GCudaLayer*)pUpStreamLayer)->deviceError(), inputStart);
		m_engine.sync();
	}
	else
	{
		if(m_incoming.size() != inputs())
			m_incoming.resize(inputs());
		m_weights.backPropError(m_engine, m_error, m_incoming, inputStart);
		m_engine.sync();
		m_incoming.download(pUpStreamLayer->error());
	}
}

// virtual
void GNeuralNetLayerCuda::updateBias(double learningRate, double momentum)
{
	m_bias.add(m_engine, m_error, learningRate);
	m_engine.sync();
}

// virtual
void GNeuralNetLayerCuda::updateWeights(const double* pUpStreamActivation, size_t inputStart, size_t inputCount, double learningRate, double momentum)
{
	// Assume that the input was already uploaded into m_incoming when feedForward was called
	if(inputStart != 0 || inputCount != m_weights.rows())
		throw Ex("Sorry, partial weight updates are not yet supported in GNeuralNetLayerCuda");
	m_weights.updateWeights(m_engine, m_incoming, m_error, learningRate);
}

// virtual
void GNeuralNetLayerCuda::updateWeights(GNeuralNetLayer* pUpStreamLayer, size_t inputStart, double learningRate, double momentum)
{
	if(inputStart != 0)
		throw Ex("Sorry, partial weight updates are not yet supported in GNeuralNetLayerCuda");
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.updateWeights(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), m_error, learningRate);
	}
	else
	{
		// Assume that the input was already uploaded into m_incoming when feedForward was called
		m_weights.updateWeights(m_engine, m_incoming, m_error, learningRate);
	}
}

void GNeuralNetLayerCuda::scaleWeights(double factor)
{
	m_weights.scale(m_engine, factor);
	m_bias.scale(m_engine, factor);
	size_t outputCount = outputs();
	m_engine.sync();
}

void GNeuralNetLayerCuda::diminishWeights(double amount)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::diminishWeights is not yet implemented");
}

// virtual
void GNeuralNetLayerCuda::clipWeights(double max)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::clipWeights is not yet implemented");
}

// virtual
size_t GNeuralNetLayerCuda::countWeights()
{
	throw Ex("Sorry, GNeuralNetLayerCuda::countWeights is not yet implemented");
	//return 0;
}

// virtual
size_t GNeuralNetLayerCuda::weightsToVector(double* pOutVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::weightsToVector is not yet implemented");
	//return 0;
}

// virtual
size_t GNeuralNetLayerCuda::vectorToWeights(const double* pVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::vectorToWeights is not yet implemented");
	//return 0;
}

// virtual
void GNeuralNetLayerCuda::copyWeights(const GNeuralNetLayer* pSource)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::copyWeights is not yet implemented");
}

void GNeuralNetLayerCuda::upload(GNeuralNetLayerClassic& source)
{
	m_weights.upload(source.weights());
	m_bias.upload(source.bias(), source.outputs());
}

void GNeuralNetLayerCuda::download(GNeuralNetLayerClassic& dest)
{
	m_weights.download(dest.weights());
	m_bias.download(dest.bias());
}


} // namespace GClasses

