#include "hip/hip_runtime.h"
/*
  The contents of this file are dedicated by all of its authors, including

    Michael S. Gashler,
    anonymous contributors,

  to the public domain (http://creativecommons.org/publicdomain/zero/1.0/).

  Note that some moral obligations still exist in the absence of legal ones.
  For example, it would still be dishonest to deliberately misrepresent the
  origin of a work. Although we impose no legal requirements to obtain a
  license, it is beseeming for those who build on the works of others to
  give back useful improvements, or find a way to pay it forward. If
  you would like to cite us, a published paper about Waffles can be found
  at http://jmlr.org/papers/volume12/gashler11a/gashler11a.pdf. If you find
  our code to be useful, the Waffles team would love to hear how you use it.
*/

#include "GCudaLayers.h"
#include "../../GClasses/GNeuralNet.h"

GCudaLayer::GCudaLayer(GDomNode* pNode)
: GNeuralNetLayer()
{
	throw Ex("Sorry, GCudaLayer does not support serialization");
}

GDomNode* GCudaLayer::serialize(GDom* pDoc)
{
	throw Ex("Sorry, GNeuralNetLayerCuda does not support serialization");
	return NULL;
}











GNeuralNetLayerCuda::GNeuralNetLayerCuda(GCudaEngine engine, size_t inputs, size_t outputs)
: GCudaLayer(engine), m_pOutgoing(NULL)
{
	resize(inputs, outputs, NULL);
}

GNeuralNetLayerCuda::~GNeuralNetLayerCuda()
{
	delete[] m_pOutgoing;
}

void GNeuralNetLayerCuda::resize(size_t inputCount, size_t outputCount, GRand* pRand)
{
	if(inputCount == inputs() && outputCount == outputs())
		return;
	if(pRand)
		throw Ex("Sorry, GNeuralNetLayerCuda does not support preserving resizes");

	m_weights.resize(inputCount, outputCount);
	m_delta.resize(inputCount, outputCount);
	m_bias.resize(6, outputCount);
	m_net.resize(6, outputCount);
	m_activation.resize(6, outputCount);
	m_error.resize(6, outputCount);
	m_biasDelta.resize(6, outputCount);
	delete[] m_pOutgoing;
	m_pOutgoing = NULL;
}

// virtual
void GNeuralNetLayerCuda::resetWeights(GRand& rand)
{
	size_t inputCount = inputs();
	size_t outputCount = outputs();
	GMatrix mTmp(inputCount, outputCount);
	double mag = 1.0 / inputCount;
	for(size_t i = 0; i < inputCount; i++)
	{
		double* pW = mTmp[i];
		for(size_t j = 0; j < inputCount; j++)
			*(pW++) = rand.normal() * mag;
	}
	m_weights.upload(mTmp);
	mTmp.setAll(0.0);
	m_delta.upload(mTmp);
	GVec vTmp(outputCount);
	double* pB = vTmp.v;
	for(size_t i = 0; i < outputCount; i++)
		*(pB++) = rand.normal() * mag;
	m_bias.upload(vTmp.v, outputCount);
	GVec::setAll(vTmp.v, 0.0, outputCount);
	m_biasDelta.upload(vTmp.v, outputCount);
}

// virtual
void GNeuralNetLayerCuda::perturbWeights(GRand& rand, double deviation, size_t start, size_t count)
{
	// Perturb weights
	GMatrix m;
	m_weights.download(m);
	size_t n = std::min(outputs() - start, count);
	for(size_t j = 0; j < m_weights.rows(); j++)
		GVec::perturb(m[j] + start, deviation, n, rand);
	m_weights.upload(m);

	// Perturb biases
	GVec v(outputCount());
	m_bias.download(v.v);
	GVec::perturb(v.v + start, deviation, n, rand);
	m_bias.upload(v.v, outputCount());
}

// virtual
double* GNeuralNetLayerCuda::activation()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputCount()];
	m_activation.download(m_pOutgoing, outputCount());
	return m_pOutgoing;
}

// virtual
double* GNeuralNetLayerCuda::error()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputCount()];
	m_error.download(m_pOutgoing, outputCount());
	return m_pOutgoing;
}

// virtual
void GNeuralNetLayerCuda::copyBiasToNet()
{
	m_activation.copy(m_bias);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

// virtual
void GNeuralNetLayerCuda::feedIn(const double* pIn, size_t inputStart, size_t inputCount)
{
	m_incoming.upload(pIn, inputCount());
	m_weights.feedIn(m_engine, m_incoming, m_activation, inputStart);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

// virtual
void GNeuralNetLayerCuda::feedIn(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.feedIn(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), m_activation, inputStart);
		if(hipDeviceSynchronize() != hipSuccess)
			throw Ex(hipGetErrorString(hipGetLastError()));
	}
	else
		feedIn(pUpStreamLayer->activation(), inputStart, pUpStreamLayer->outputs());
}

// virtual
void GNeuralNetLayerCuda::activate()
{
	m_activation.activateTanh(m_engine);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::computeError(const double* pTarget)
{
	m_error.upload(pTarget, outputCount());
	m_error.add(m_engine, m_activation, -1.0);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::deactivateError()
{
	m_error.deactivateError(m_activation);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::backPropError(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.backPropError(m_error, ((GCudaLayer*)pUpStreamLayer)->deviceError(), inputStart);
		if(hipDeviceSynchronize() != hipSuccess)
			throw Ex(hipGetErrorString(hipGetLastError()));
	}
	else
	{
		if(m_incoming.size() != inputCount())
			m_incoming.resize(inputCount());
		m_weights.backPropError(m_error, m_incoming, inputStart);
		if(hipDeviceSynchronize() != hipSuccess)
			throw Ex(hipGetErrorString(hipGetLastError()));
		m_incoming.download(pUpStreamLayer->error(), inputCount());
	}
}

// virtual
void GNeuralNetLayerCuda::adjustWeights(const double* pUpStreamActivation, double learningRate, double momentum)
{
	// Assume that the input was already uploaded into m_incoming when feedForward was called
	m_weights.updateWeights(m_engine, m_incoming, m_error, learningRate);
	m_bias.add(m_engine, m_error, learningRate);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

// virtual
void GNeuralNetLayerCuda::adjustWeights(GNeuralNetLayer* pUpStreamLayer, double learningRate, double momentum)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.updateWeights(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), m_error, learningRate);
		m_bias.add(m_engine, m_error, learningRate);
	}
	else
	{
		// Assume that the input was already uploaded into m_incoming when feedForward was called
		m_weights.updateWeights(m_engine, m_incoming, m_error, learningRate);
		m_bias.add(m_engine, m_error, learningRate);
	}
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::scaleWeights(double factor)
{
	m_weights.scale(factor);
	m_bias.scale(factor);
	size_t outputCount = outputs();
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::diminishWeights(double amount)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::diminishWeights is not yet implemented");
}

// virtual
void GNeuralNetLayerCuda::clipWeights(double max)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::clipWeights is not yet implemented");
}

// virtual
size_t GNeuralNetLayerCuda::countWeights()
{
	throw Ex("Sorry, GNeuralNetLayerCuda::countWeights is not yet implemented");
	return 0;
}

// virtual
size_t GNeuralNetLayerCuda::weightsToVector(double* pOutVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::weightsToVector is not yet implemented");
	return 0;
}

// virtual
size_t GNeuralNetLayerCuda::vectorToWeights(const double* pVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::vectorToWeights is not yet implemented");
	return 0;
}

// virtual
void GNeuralNetLayerCuda::copyWeights(const GNeuralNetLayer* pSource)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::copyWeights is not yet implemented");
	return 0;
}

