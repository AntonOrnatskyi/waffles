#include "hip/hip_runtime.h"
/*
  The contents of this file are dedicated by all of its authors, including

    Michael S. Gashler,
    anonymous contributors,

  to the public domain (http://creativecommons.org/publicdomain/zero/1.0/).

  Note that some moral obligations still exist in the absence of legal ones.
  For example, it would still be dishonest to deliberately misrepresent the
  origin of a work. Although we impose no legal requirements to obtain a
  license, it is beseeming for those who build on the works of others to
  give back useful improvements, or find a way to pay it forward. If
  you would like to cite us, a published paper about Waffles can be found
  at http://jmlr.org/papers/volume12/gashler11a/gashler11a.pdf. If you find
  our code to be useful, the Waffles team would love to hear how you use it.
*/

#include "GCudaLayers.h"
#include "../../GClasses/GNeuralNet.h"
#include "../../GClasses/GVec.h"

namespace GClasses {

GCudaLayer::GCudaLayer(GDomNode* pNode, GCudaEngine& engine)
: GNeuralNetLayer(), m_engine(engine)
{
	throw Ex("Sorry, GCudaLayer does not support serialization");
}

GDomNode* GCudaLayer::serialize(GDom* pDoc)
{
	throw Ex("Sorry, GNeuralNetLayerCuda does not support serialization");
	//return NULL;
}











GNeuralNetLayerCuda::GNeuralNetLayerCuda(GCudaEngine engine, size_t inputs, size_t outputs)
: GCudaLayer(engine), m_pOutgoing(NULL)
{
	resize(inputs, outputs, NULL);
}

GNeuralNetLayerCuda::~GNeuralNetLayerCuda()
{
	delete[] m_pOutgoing;
}

void GNeuralNetLayerCuda::resize(size_t inputCount, size_t outputCount, GRand* pRand)
{
	if(inputCount == inputs() && outputCount == outputs())
		return;
	if(pRand)
		throw Ex("Sorry, GNeuralNetLayerCuda does not support preserving resizes");

	m_weights.resize(inputCount, outputCount);
	m_bias.resize(outputCount);
	m_activation.resize(outputCount);
	m_error.resize(outputCount);
	delete[] m_pOutgoing;
	m_pOutgoing = NULL;
}

// virtual
void GNeuralNetLayerCuda::resetWeights(GRand& rand)
{
	size_t inputCount = inputs();
	size_t outputCount = outputs();
	GMatrix mTmp(inputCount, outputCount);
	double mag = 1.0 / inputCount;
	for(size_t i = 0; i < inputCount; i++)
	{
		double* pW = mTmp[i];
		for(size_t j = 0; j < inputCount; j++)
			*(pW++) = rand.normal() * mag;
	}
	m_weights.upload(mTmp);
	GVec vTmp(outputCount);
	double* pB = vTmp.v;
	for(size_t i = 0; i < outputCount; i++)
		*(pB++) = rand.normal() * mag;
	m_bias.upload(vTmp.v, outputCount);
}

// virtual
void GNeuralNetLayerCuda::perturbWeights(GRand& rand, double deviation, size_t start, size_t count)
{
	// Perturb weights
	GMatrix m;
	m_weights.download(m);
	size_t n = std::min(outputs() - start, count);
	for(size_t j = 0; j < m_weights.rows(); j++)
		GVec::perturb(m[j] + start, deviation, n, rand);
	m_weights.upload(m);

	// Perturb biases
	GVec v(outputs());
	m_bias.download(v.v);
	GVec::perturb(v.v + start, deviation, n, rand);
	m_bias.upload(v.v, outputs());
}

// virtual
double* GNeuralNetLayerCuda::activation()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputs()];
	m_activation.download(m_pOutgoing);
	return m_pOutgoing;
}

// virtual
double* GNeuralNetLayerCuda::error()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputs()];
	m_error.download(m_pOutgoing);
	return m_pOutgoing;
}

// virtual
void GNeuralNetLayerCuda::copyBiasToNet()
{
	m_activation.copy(m_engine, m_bias);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

// virtual
void GNeuralNetLayerCuda::feedIn(const double* pIn, size_t inputStart, size_t inputCount)
{
	m_incoming.upload(pIn, inputs());
	m_weights.feedIn(m_engine, m_incoming, m_activation, inputStart);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

// virtual
void GNeuralNetLayerCuda::feedIn(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.feedIn(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), m_activation, inputStart);
		if(hipDeviceSynchronize() != hipSuccess)
			throw Ex(hipGetErrorString(hipGetLastError()));
	}
	else
		feedIn(pUpStreamLayer->activation(), inputStart, pUpStreamLayer->outputs());
}

// virtual
void GNeuralNetLayerCuda::activate()
{
	m_activation.activateTanh(m_engine);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::computeError(const double* pTarget)
{
	m_error.upload(pTarget, outputs());
	m_error.add(m_engine, m_activation, -1.0);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::deactivateError()
{
	m_error.deactivateTanh(m_engine, m_activation);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::backPropError(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.backPropError(m_engine, m_error, ((GCudaLayer*)pUpStreamLayer)->deviceError(), inputStart);
		if(hipDeviceSynchronize() != hipSuccess)
			throw Ex(hipGetErrorString(hipGetLastError()));
	}
	else
	{
		if(m_incoming.size() != inputs())
			m_incoming.resize(inputs());
		m_weights.backPropError(m_engine, m_error, m_incoming, inputStart);
		if(hipDeviceSynchronize() != hipSuccess)
			throw Ex(hipGetErrorString(hipGetLastError()));
		m_incoming.download(pUpStreamLayer->error());
	}
}

// virtual
void GNeuralNetLayerCuda::adjustWeights(const double* pUpStreamActivation, double learningRate, double momentum)
{
	// Assume that the input was already uploaded into m_incoming when feedForward was called
	m_weights.updateWeights(m_engine, m_incoming, m_error, learningRate);
	m_bias.add(m_engine, m_error, learningRate);
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

// virtual
void GNeuralNetLayerCuda::adjustWeights(GNeuralNetLayer* pUpStreamLayer, double learningRate, double momentum)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.updateWeights(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), m_error, learningRate);
		m_bias.add(m_engine, m_error, learningRate);
	}
	else
	{
		// Assume that the input was already uploaded into m_incoming when feedForward was called
		m_weights.updateWeights(m_engine, m_incoming, m_error, learningRate);
		m_bias.add(m_engine, m_error, learningRate);
	}
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::scaleWeights(double factor)
{
	m_weights.scale(m_engine, factor);
	m_bias.scale(m_engine, factor);
	size_t outputCount = outputs();
	if(hipDeviceSynchronize() != hipSuccess)
		throw Ex(hipGetErrorString(hipGetLastError()));
}

void GNeuralNetLayerCuda::diminishWeights(double amount)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::diminishWeights is not yet implemented");
}

// virtual
void GNeuralNetLayerCuda::clipWeights(double max)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::clipWeights is not yet implemented");
}

// virtual
size_t GNeuralNetLayerCuda::countWeights()
{
	throw Ex("Sorry, GNeuralNetLayerCuda::countWeights is not yet implemented");
	//return 0;
}

// virtual
size_t GNeuralNetLayerCuda::weightsToVector(double* pOutVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::weightsToVector is not yet implemented");
	//return 0;
}

// virtual
size_t GNeuralNetLayerCuda::vectorToWeights(const double* pVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::vectorToWeights is not yet implemented");
	//return 0;
}

// virtual
void GNeuralNetLayerCuda::copyWeights(const GNeuralNetLayer* pSource)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::copyWeights is not yet implemented");
}

} // namespace GClasses

