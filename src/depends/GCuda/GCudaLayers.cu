#include "hip/hip_runtime.h"
/*
  The contents of this file are dedicated by all of its authors, including

    Michael S. Gashler,
    anonymous contributors,

  to the public domain (http://creativecommons.org/publicdomain/zero/1.0/).

  Note that some moral obligations still exist in the absence of legal ones.
  For example, it would still be dishonest to deliberately misrepresent the
  origin of a work. Although we impose no legal requirements to obtain a
  license, it is beseeming for those who build on the works of others to
  give back useful improvements, or find a way to pay it forward. If
  you would like to cite us, a published paper about Waffles can be found
  at http://jmlr.org/papers/volume12/gashler11a/gashler11a.pdf. If you find
  our code to be useful, the Waffles team would love to hear how you use it.
*/

#include "GCudaLayers.h"
#include "../../GClasses/GNeuralNet.h"
#include "../../GClasses/GVec.h"

namespace GClasses {

GCudaLayer::GCudaLayer(GDomNode* pNode, GCudaEngine& engine)
: GNeuralNetLayer(), m_engine(engine)
{
	throw Ex("Sorry, GCudaLayer does not support serialization");
}

GDomNode* GCudaLayer::serialize(GDom* pDoc)
{
	throw Ex("Sorry, GNeuralNetLayerCuda does not support serialization");
	//return NULL;
}











GLayerCuda::GLayerCuda(GCudaEngine& engine, size_t inputs, size_t outputs)
: GCudaLayer(engine), m_pOutgoing(NULL)
{
	resize(inputs, outputs, NULL);
}

GLayerCuda::~GLayerCuda()
{
	delete[] m_pOutgoing;
}

void GLayerCuda::resize(size_t inputCount, size_t outputCount, GRand* pRand)
{
	if(inputCount == inputs() && outputCount == outputs())
		return;
	if(pRand)
		throw Ex("Sorry, GLayerCuda does not support preserving resizes");

	m_weights.resize(inputCount, outputCount);
	m_bias.resize(outputCount);
	m_activation.resize(outputCount);
	m_error.resize(outputCount);
	delete[] m_pOutgoing;
	m_pOutgoing = NULL;
}

// virtual
void GLayerCuda::resetWeights(GRand& rand)
{
	size_t inputCount = inputs();
	size_t outputCount = outputs();
	GMatrix mTmp(inputCount, outputCount);
	double mag = 1.0 / inputCount;
	for(size_t i = 0; i < inputCount; i++)
	{
		double* pW = mTmp[i];
		for(size_t j = 0; j < outputCount; j++)
			*(pW++) = rand.normal() * mag;
	}
	m_weights.upload(mTmp);
	GVec vTmp(outputCount);
	double* pB = vTmp.v;
	for(size_t i = 0; i < outputCount; i++)
		*(pB++) = rand.normal() * mag;
	m_bias.upload(vTmp.v, outputCount);
}

// virtual
void GLayerCuda::perturbWeights(GRand& rand, double deviation, size_t start, size_t count)
{
	// Perturb weights
	GMatrix m;
	m_weights.download(m);
	size_t n = std::min(outputs() - start, count);
	for(size_t j = 0; j < m_weights.rows(); j++)
		GVec::perturb(m[j] + start, deviation, n, rand);
	m_weights.upload(m);

	// Perturb biases
	GVec v(outputs());
	m_bias.download(v.v);
	GVec::perturb(v.v + start, deviation, n, rand);
	m_bias.upload(v.v, outputs());
}

// virtual
double* GLayerCuda::activation()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputs()];
	m_activation.download(m_pOutgoing);
	return m_pOutgoing;
}

// virtual
double* GLayerCuda::error()
{
	if(!m_pOutgoing)
		m_pOutgoing = new double[outputs()];
	m_error.download(m_pOutgoing);
	return m_pOutgoing;
}

// virtual
void GLayerCuda::copyBiasToNet()
{
	m_activation.copy(m_engine, m_bias);
	m_engine.sync();
}

// virtual
void GLayerCuda::feedIn(const double* pIn, size_t inputStart, size_t inputCount)
{
	m_incoming.upload(pIn, inputs());
	m_weights.feedIn(m_engine, m_incoming, m_activation, inputStart);
	m_engine.sync();
}

// virtual
void GLayerCuda::feedIn(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.feedIn(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), m_activation, inputStart);
		m_engine.sync();
	}
	else
		feedIn(pUpStreamLayer->activation(), inputStart, pUpStreamLayer->outputs());
}

// virtual
void GLayerCuda::activate()
{
	m_activation.activateTanh(m_engine);
	m_engine.sync();
}

// virtual
void GLayerCuda::dropOut(GRand& rand, double probOfDrop)
{
	throw Ex("sorry, not implemented yet");
}

// virtual
void GLayerCuda::dropConnect(GRand& rand, double probOfDrop)
{
	throw Ex("sorry, not implemented yet");
}

void GLayerCuda::computeError(const double* pTarget)
{
	m_error.upload(pTarget, outputs());
	m_error.add(m_engine, m_activation, -1.0);
	m_engine.sync();
}

void GLayerCuda::deactivateError()
{
	m_error.deactivateTanh(m_engine, m_activation);
	m_engine.sync();
}

void GLayerCuda::backPropError(GNeuralNetLayer* pUpStreamLayer, size_t inputStart)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.backPropError(m_engine, m_error, ((GCudaLayer*)pUpStreamLayer)->deviceError(), inputStart);
		m_engine.sync();
	}
	else
	{
		if(m_incoming.size() != inputs())
			m_incoming.resize(inputs());
		m_weights.backPropError(m_engine, m_error, m_incoming, inputStart);
		m_engine.sync();
		m_incoming.download(pUpStreamLayer->error());
	}
}

// virtual
void GLayerCuda::updateBias(double learningRate, double momentum)
{
	m_bias.add(m_engine, m_error, learningRate);
	m_engine.sync();
}

// virtual
void GLayerCuda::updateWeights(const double* pUpStreamActivation, size_t inputStart, size_t inputCount, double learningRate, double momentum)
{
	// Assume that the input was already uploaded into m_incoming when feedForward was called
	if(inputStart != 0 || inputCount != m_weights.rows())
		throw Ex("Sorry, partial weight updates are not yet supported in GNeuralNetLayerCuda");
	m_weights.updateWeights(m_engine, m_incoming, inputStart, m_error, learningRate);
}

// virtual
void GLayerCuda::updateWeights(GNeuralNetLayer* pUpStreamLayer, size_t inputStart, double learningRate, double momentum)
{
	if(pUpStreamLayer->usesGPU())
	{
		m_weights.updateWeights(m_engine, ((GCudaLayer*)pUpStreamLayer)->deviceActivation(), inputStart, m_error, learningRate);
	}
	else
	{
		// Assume that the input was already uploaded into m_incoming when feedForward was called
		if(inputStart != 0)
			throw Ex("Sorry, partial weight updates are not yet supported in GNeuralNetLayerCuda");
		m_weights.updateWeights(m_engine, m_incoming, inputStart, m_error, learningRate);
	}
}

// virtual
void GLayerCuda::updateWeightsAndRestoreDroppedOnes(const double* pUpStreamActivation, size_t inputStart, size_t inputCount, double learningRate, double momentum)
{
	throw Ex("Sorry, not implemented yet");
}

// virtual
void GLayerCuda::updateWeightsAndRestoreDroppedOnes(GNeuralNetLayer* pUpStreamLayer, size_t inputStart, double learningRate, double momentum)
{
	throw Ex("Sorry, not implemented yet");
}

void GLayerCuda::scaleWeights(double factor, bool scaleBiases)
{
	m_weights.scale(m_engine, factor);
	if(scaleBiases)
		m_bias.scale(m_engine, factor);
	m_engine.sync();
}

void GLayerCuda::diminishWeights(double amount, bool diminishBiases)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::diminishWeights is not yet implemented");
}

// virtual
void GLayerCuda::maxNorm(double max)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::maxNorm is not yet implemented");
}

// virtual
double GLayerCuda::unitIncomingWeightsL1Norm(size_t unit)
{
	return m_weights.colSumAbs(m_engine, unit);
}

// virtual
double GLayerCuda::unitIncomingWeightsL2Norm(size_t unit)
{
	return m_weights.colSumSquare(m_engine, unit);
}

// virtual
double GLayerCuda::unitOutgoingWeightsL1Norm(size_t input)
{
	return m_weights.rowSumAbs(m_engine, input);
}

// virtual
double GLayerCuda::unitOutgoingWeightsL2Norm(size_t input)
{
	return m_weights.rowSumSquare(m_engine, input);
}

// virtual
void GLayerCuda::scaleUnitIncomingWeights(size_t unit, double scalar)
{
	m_weights.scaleCol(m_engine, unit, scalar);
}

// virtual
void GLayerCuda::scaleUnitOutgoingWeights(size_t input, double scalar)
{
	m_weights.scaleRow(m_engine, input, scalar);
}

// virtual
size_t GLayerCuda::countWeights()
{
	throw Ex("Sorry, GNeuralNetLayerCuda::countWeights is not yet implemented");
	//return 0;
}

// virtual
size_t GLayerCuda::weightsToVector(double* pOutVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::weightsToVector is not yet implemented");
	//return 0;
}

// virtual
size_t GLayerCuda::vectorToWeights(const double* pVector)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::vectorToWeights is not yet implemented");
	//return 0;
}

// virtual
void GLayerCuda::copyWeights(GNeuralNetLayer* pSource)
{
	throw Ex("Sorry, GNeuralNetLayerCuda::copyWeights is not yet implemented");
}

void GLayerCuda::upload(GLayerClassic& source)
{
	m_weights.upload(source.weights());
	m_bias.upload(source.bias(), source.outputs());
}

void GLayerCuda::download(GLayerClassic& dest)
{
	m_weights.download(dest.weights());
	m_bias.download(dest.bias());
}


} // namespace GClasses

